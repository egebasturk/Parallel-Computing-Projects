#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils.cuh"
#include "kernels.cu"
#define DEBUG_STOP

// Reference: Print device properties codes were taken from
// http://www.cs.fsu.edu/~xyuan/cda5125/examples/lect24/devicequery.cu

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %u\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %u\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %u\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %u\n",  devProp.totalConstMem);
    printf("Texture alignment:             %u\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}
/*
Arguments
1. The number of threads used to compute Matrix-vector product
2. The number of repetitions and
3. An argument to print on stdout (See below).
4. Test-file name
*/
int main(int argc, char* argv[]) {
    // Number of CUDA devices
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);
 
    // Iterate through devices
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }
    getchar();
    // Matrix meta-data
    int rows, columns, num_of_non_zero_entries;
    // Matrix
    int* row_ptr_array, *col_ind_array;
    double* values_array, *x_array;
    // Matrix on device
    int* row_ptr_array_d, *col_ind_array_d;
    double* values_array_d, *x_array_d;

    int num_threads         = atoi(argv[1]);
    int num_repetitions     = atoi(argv[2]);
    int flag_stdout         = atoi(argv[3]);
    char* input_filename    = argv[4];

    readMatrixFromFile(input_filename,
                        // First row of file
                       &rows, &columns, &num_of_non_zero_entries,
                        // Return variables
                       &row_ptr_array, &col_ind_array, &values_array);
    // Init. x to 1 (in kernel)
    x_array = (double*)malloc(sizeof(double) * rows);
    for (int i = 0; i < rows; i++)
    {
        x_array[i] = 1.0f;
    }
    if (flag_stdout == 1)
    {
        printf("Input Matrix:\n");
        printMatrix(rows, columns, num_of_non_zero_entries,
                row_ptr_array, col_ind_array, values_array);
        #ifdef DEBUG_STOP
        getchar();
        #endif
        printf("Initial Vector:\n");
        printVector(rows, x_array);
        #ifdef DEBUG_STOP
        getchar();
        #endif
    }
    size_t size = num_of_non_zero_entries * sizeof(int) +
        num_of_non_zero_entries * sizeof(int) +
        num_of_non_zero_entries * sizeof(double) +
        rows * sizeof(double);
    //hipDeviceSetLimit(hipLimitMallocHeapSize, size);

    // Allocate on device
    hipMalloc(&row_ptr_array_d, num_of_non_zero_entries * sizeof(int));
    hipMalloc(&col_ind_array_d, num_of_non_zero_entries * sizeof(int));
    hipMalloc(&values_array_d, num_of_non_zero_entries * sizeof(double));
    hipMalloc(&x_array_d, rows * sizeof(double));
    CUDAErrorCheck("Malloc Error");
    #ifdef DEBUG_STOP
    getchar();
    #endif
    // Copy
    hipMemcpy(row_ptr_array_d, row_ptr_array,
        num_of_non_zero_entries * sizeof(int), hipMemcpyHostToDevice);
        
    hipMemcpy(col_ind_array_d, col_ind_array,
        num_of_non_zero_entries * sizeof(int), hipMemcpyHostToDevice);
        
    hipMemcpy(values_array_d, values_array,
        num_of_non_zero_entries * sizeof(double), hipMemcpyHostToDevice);
        
    hipMemcpy(x_array_d, x_array,
        rows * sizeof(double), hipMemcpyHostToDevice);

    CUDAErrorCheck("Memcpy Error");
    #ifdef DEBUG_STOP
    getchar();
    #endif
        
    // Kernel invocation here
    int tmp = ceil(rows / num_threads);
    dim3 dimGrid(tmp,1);
    dim3 dimBlock(num_threads, 1);
    getchar();
    mmult_kernel<<<dimGrid, dimBlock>>>(rows, columns, num_of_non_zero_entries,
                                        num_repetitions,
                                        row_ptr_array_d, col_ind_array_d,
                                        values_array_d, x_array_d );
    CUDAErrorCheck("Kernel Error");
    #ifdef DEBUG_STOP
    getchar();
    #endif
    // Read back from the device
//    hipMemcpy(row_ptr_array, row_ptr_array_d, rows, hipMemcpyDeviceToHost);
//    hipMemcpy(col_ind_array, col_ind_array_d, columns, hipMemcpyDeviceToHost);
//    hipMemcpy(values_array, values_array_d, rows, hipMemcpyDeviceToHost);
    x_array[0] = 6666;
    hipMemcpy(x_array, x_array_d,
        rows * sizeof(double), hipMemcpyDeviceToHost);
    CUDAErrorCheck("Memcpy back error");
    #ifdef DEBUG_STOP
    getchar();
    #endif
    if (flag_stdout == 1 || flag_stdout == 2)
    {
        printf("Resulting Vector:\n");
        printVector(rows, x_array);
    }
    hipFree(x_array_d);
    hipFree(row_ptr_array_d);
    hipFree(col_ind_array_d);
    hipFree(values_array_d);
    free(x_array);
    free(row_ptr_array);
    free(col_ind_array);
    free(values_array);
    return 0;
}
