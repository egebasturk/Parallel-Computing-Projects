#include "utils.cuh"
__host__
void readMatrixFromFile(char* input_filename,
                        // First row of file
                        int* rows, int* columns, int* num_of_non_zero_entries,
                        // Return variables
                        int** row_ptr_array, int** row_ptr_array_init,
                        int** col_ind_array,                        
                        double** values_array) {
    FILE *fptr;

    fptr = fopen(input_filename, "r");
    if (fptr == NULL) {
        printf("Error reading file");
        return;
    } else {
        int index = 0;
        int row = 0, column = 0, tmp = 0;
        double non_zero_val = 0.0;
        // Read first row from matrix file
        fscanf(fptr, "%d %d %d", &row, &column, &tmp);
        *rows = row;
        *columns = column;
        *num_of_non_zero_entries = tmp;
        int* elem_ptr_arr = (int*)malloc(sizeof(int) * (*rows + 1));
        *row_ptr_array = (int*)malloc(sizeof(int) * (*rows + 1));
        *row_ptr_array_init = (int*)malloc(sizeof(int) * *num_of_non_zero_entries);
        *col_ind_array = (int*)malloc(sizeof(int) * *num_of_non_zero_entries);
        *values_array  = (double*)malloc(sizeof(double) * *num_of_non_zero_entries);
        
        
        // read lines into 3 variables line by line
        while (index < *num_of_non_zero_entries)
        {
            fscanf(fptr, "%d", &row);
            fscanf(fptr, "%d", &column);
            fscanf(fptr, "%lf", &non_zero_val);
        
            // -1 to make indices start from 0
            elem_ptr_arr[row - 1]++;
//            (*row_ptr_array)[index] = row - 1;
            (*row_ptr_array_init)[index] = row - 1;
            (*col_ind_array)[index] = column - 1;
            (*values_array)[index]  = non_zero_val;
            index++;
        }
        // Prefix sum
        for (int k = 1; k < *rows; k++)
        {
            elem_ptr_arr[k] += elem_ptr_arr[k - 1];
            (*row_ptr_array)[k] = elem_ptr_arr[k];
        }
//        for (int k = *rows; k > 0; k--)
//            (*row_ptr_array)[k] = *num_of_non_zero_entries - elem_ptr_arr[k - 1];
        (*row_ptr_array)[0] = 0;
        free(elem_ptr_arr);
    }
}
__host__
void printMatrix(int rows, int columns, int num_of_non_zero_entries,
                int* row_ptr_array, int* col_ind_array,
                double * values_array) {
    printf("%d\t%d\t%d\n", rows, columns, num_of_non_zero_entries);
    // For each row
    for (int i = 0; i < num_of_non_zero_entries; i++)
    {
        printf("%d\t%d\t%lf\n", row_ptr_array[i] + 1,
            col_ind_array[i] + 1, values_array[i]);
    }
}
__host__
void printVector(int rows, double* x_array)
{
    for (int i = 0; i < rows; i++)
    {
        printf("%lf\n", x_array[i]);
    }
    printf("\n");
}
__host__
void CUDAErrorCheck(const char* msg) {
hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %d: %s.\n", msg, (int)err, hipGetErrorName(err));
//    exit(EXIT_FAILURE);
  }
}
__host__
void mmult_serial(// First row of file
                       int rows, int columns, int num_of_non_zero_entries,
                       int num_repetitions,
                        // Return variables
                       int* row_ptr_array, int* col_ind_array,
                       double* values_array, double** x_array, double** x_array_old)
{
    for ( int row = 0; row < rows; row++)
    {
        double tmp_product = 0;
        int row_start = row_ptr_array[row];
        int row_end   = row_ptr_array[row + 1];
        
        // Iterate over the sparse row
        for (int j = row_start; j < row_end; j++)
            tmp_product += values_array[j] * (*x_array_old)[col_ind_array[j]];
        (*x_array)[row] = tmp_product;
    }
}
/*__host__
void init_vector_to_1(// First row of file
                       int rows, int columns, int num_of_non_zero_entries,
                       int num_repetitions,
                        // Return variables
                       int* row_ptr_array, int* col_ind_array,
                       double* values_array, double** x_array)
{
    for ( int row = 0; row < rows; row++)
    {
        int row_start = row_ptr_array[row];
        int row_end   = row_ptr_array[row + 1];
        
        // Iterate over the sparse row
        for (int j = row_start; j < row_end; j++)
            (*x_array)[col_ind_array[j]] = 1.0f;
    }
}*/
